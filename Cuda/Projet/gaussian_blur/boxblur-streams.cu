#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <string>
#include <chrono>

__global__ void grayscale_boxblur_shared(unsigned char* rgb, unsigned char* s, std::size_t cols, std::size_t rows)
{
    auto i = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
    auto j = blockIdx.y * (blockDim.y - 2) + threadIdx.y;

    auto li = threadIdx.x;
    auto lj = threadIdx.y;

    auto w = blockDim.x;
    auto h = blockDim.y;

    extern __shared__ unsigned char sh[];

    if (i < cols && j < rows) {
        sh[lj * w + li] = (307 * rgb[3 * (j * cols + i)] + 604 * rgb[3 * (j * cols + i) + 1] + 113 * rgb[3 * (j * cols + i) + 2]) >> 10;
    }

    __syncthreads();

    if (i < cols - 1 && j < rows - 1 && li > 0 && li < (w - 1) && lj > 0 && lj < (h - 1))
    {
        auto total = sh[((lj - 1) * w + li - 1)] + sh[((lj - 1) * w + li)] + sh[((lj - 1) * w + li + 1)]
                     + sh[(lj * w + li - 1)] + sh[(lj * w + li)] + sh[(lj * w + li + 1)]
                     + sh[((lj + 1) * w + li - 1)] + sh[((lj + 1) * w + li)] + sh[((lj + 1) * w + li + 1)];

        auto res = total / 9;
        s[j * cols + i] = res;
    }
}

int main() {
    // Lecture de l'image d'entrée
    cv::Mat m_in = cv::imread("../images/in.jpg", cv::IMREAD_UNCHANGED);
    auto rows = m_in.rows;
    auto cols = m_in.cols;
    auto start = std::chrono::high_resolution_clock::now();


    // Allocation et copie des données sur le GPU
    unsigned char* rgb_d = nullptr;
    unsigned char* g_d = nullptr;
    unsigned char* s_d = nullptr;
    hipMalloc(&rgb_d, 3 * rows * cols);
    hipMalloc(&g_d, rows * cols);
    hipMalloc(&s_d, rows * cols);
    hipMemcpy(rgb_d, m_in.data, 3 * rows * cols, hipMemcpyHostToDevice);

    // Définition des paramètres de grille et de bloc pour les kernels
    dim3 block(64, 8);
    dim3 grid0((cols - 1) / block.x + 1, (rows - 1) / block.y + 1);
    dim3 grid1((cols - 1) / (block.x - 2) + 1, (rows - 1) / (block.y - 2) + 1);

    // Création des streams CUDA
    hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    std::size_t const sizeb = (cols*rows) * sizeof( int );

    // Appel du premier kernel
    //grayscale_boxblur_shared<<<grid1, block, block.x * block.y * sizeof(unsigned char), stream[0]>>>(rgb_d, s_d, cols, rows/2);

    // Appel du deuxième kernel
    grayscale_boxblur_shared<<<grid1, block, block.x * block.y * sizeof(unsigned char), stream[1]>>>(rgb_d+(rows*cols)/2, g_d+sizeb/2, cols, rows/2);

    // Copie du résultat final sur le CPU
    unsigned char* out = nullptr;
    hipHostMalloc(&out, rows * cols);
    //hipMemcpyAsync(out, s_d, (rows * cols)/2, hipMemcpyDeviceToHost, stream[0]);
    hipMemcpyAsync(out, s_d, (rows * cols)/2, hipMemcpyDeviceToHost, stream[0]);
    hipMemcpyAsync(out+(rows * cols)/2, g_d, (rows * cols)/2, hipMemcpyDeviceToHost, stream[1]);
    //hipMemcpyAsync(out+(rows * cols)/2, g_d, (rows * cols)/2, hipMemcpyDeviceToHost, stream[1]);
    //hipMemcpyAsync(out+(rows * cols)/2, s_d+(rows * cols)/2, (rows * cols)/2, hipMemcpyDeviceToHost, stream[1]);

    cv::Mat m_out( rows, cols, CV_8UC1, out );

    // Affichage du temps d'exécution
    hipDeviceSynchronize();

    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Execution time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;
    cv::imwrite( "out.jpg", m_out );
    // Libération de la mémoire
    hipFree(rgb_d);
    hipFree(g_d);
    hipFree(s_d);
    hipHostFree(out);
    hipStreamDestroy(stream[0]);
    hipStreamDestroy(stream[1]);

    return 0;
}