#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <string>
#include <chrono>

__global__ void grayscale_laplacian_gaussian_shared( unsigned char * rgb, unsigned char * s, std::size_t cols, std::size_t rows ) {
    auto i = blockIdx.x * (blockDim.x-5) + threadIdx.x;
    auto j = blockIdx.y * (blockDim.y-5) + threadIdx.y;

    auto li = threadIdx.x;
    auto lj = threadIdx.y;

    auto w = blockDim.x;
    auto h = blockDim.y;

    extern __shared__ unsigned char sh[];

    if( i < cols && j < rows ) {
        sh[ lj * w + li ] = (
                307 * rgb[ 3 * ( j * cols + i ) ]
                + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
                + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
        ) >> 10;
    }

    /**
     * Il faut synchroniser tous les warps (threads) du bloc pour être certain que le niveau de gris est calculé
     * par tous les threads du bloc avant de pouvoir accéder aux données des pixels voisins.
     */
    __syncthreads();

    if( i < cols -2 && j < rows-2 && li > 2 && li < (w-2) && lj > 2 && lj < (h-2) )
    {
        auto res =       sh[((lj - 2) * w + li - 2) ] * 0 + sh[((lj - 2) * w + li -1) ] * 0 +  sh[((lj - 2) * w + li) ]* -1 + sh[((lj - 2) * w + li +1 ) ] * 0 + sh[((lj - 2) * w + li + 2) ] *0
                         +  sh[((lj - 1) * w + li - 2) ] * 0 + sh[((lj - 1) * w + li -1) ] * -1 +  sh[((lj - 1) * w + li) ]* -2 + sh[((lj - 1) * w + li +1 ) ] * -1 + sh[((lj - 1) * w + li + 2) ] *0
                         +     sh[((lj) * w + li - 2) ] * -1 + sh[((lj) * w + li -1) ] * -2 +  sh[((lj) * w + li) ]* 16 + sh[((lj) * w + li +1 ) ] * -2 + sh[((lj) * w + li + 2) ] * -1
                         +    sh[((lj + 1) * w + li - 2) ] * 0 + sh[((lj +1) * w + li -1) ] * -1 +  sh[((lj + 1) * w + li) ]* -2 + sh[((lj + 1) * w + li +1 ) ] * -1 + sh[((lj + 1) * w + li + 2) ] *0
                         +    sh[((lj + 2) * w + li - 2) ] * 0 + sh[((lj + 2) * w + li -1) ] * 0 +  sh[((lj + 2) * w + li) ]* -1 + sh[((lj + 2) * w + li +1 ) ] * 0 + sh[((lj + 2) * w + li + 2) ] *0;

        res = res > 255 ? 255 : res;
        res = res < 0 ? 0 : res;

        s[j * cols + i] = res;
    }
}

int main() {
    // Lecture de l'image d'entrée
    cv::Mat m_in = cv::imread("../images/in.jpg", cv::IMREAD_UNCHANGED);
    auto rows = m_in.rows;
    auto cols = m_in.cols;
    auto start = std::chrono::high_resolution_clock::now();


    // Allocation et copie des données sur le GPU
    unsigned char* rgb_d = nullptr;
    unsigned char* g_d = nullptr;
    unsigned char* s_d = nullptr;
    hipMalloc(&rgb_d, 3 * rows * cols);
    hipMalloc(&g_d, rows * cols);
    hipMalloc(&s_d, rows * cols);
    hipMemcpy(rgb_d, m_in.data, 3 * rows * cols, hipMemcpyHostToDevice);

    // Définition des paramètres de grille et de bloc pour les kernels
    dim3 block(64, 8);
    //dim3 grid0((cols - 1) / block.x + 1, (rows - 1) / block.y + 1);
    dim3 grid1((cols - 1) / (block.x - 5) + 1, (rows - 1) / (block.y - 5) + 1);

    // Création des streams CUDA
    hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    // Appel du premier kernel
    grayscale_laplacian_gaussian_shared<<<grid1, block, block.x * (block.y+2) * sizeof(unsigned char), stream[0]>>>(rgb_d, s_d, cols, rows/2+2);

    // Appel du deuxième kernel
    grayscale_laplacian_gaussian_shared<<<grid1, block, block.x * (block.y+2) * sizeof(unsigned char), stream[1]>>>(rgb_d+(((rows*cols*3)/2)-cols*3*3), g_d, cols, rows/2+3);

    // Copie du résultat final sur le CPU
    unsigned char* out = nullptr;
    hipHostMalloc(&out, rows * cols);

    hipMemcpyAsync(out, s_d, (rows * cols)/2, hipMemcpyDeviceToHost, stream[0]);
    hipMemcpyAsync(out+(rows * cols)/2, g_d+cols*3, (rows * cols)/2, hipMemcpyDeviceToHost, stream[1]);


    cv::Mat m_out( rows, cols, CV_8UC1, out );

    // Affichage du temps d'exécution
    hipDeviceSynchronize();

    //hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Execution time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;
    cv::imwrite( "out.jpg", m_out );
    // Libération de la mémoire
    hipFree(rgb_d);
    hipFree(g_d);
    hipFree(s_d);
    hipHostFree(out);
    hipStreamDestroy(stream[0]);
    hipStreamDestroy(stream[1]);

    return 0;
}