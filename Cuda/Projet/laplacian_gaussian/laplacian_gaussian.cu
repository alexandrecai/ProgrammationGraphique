#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>

/**
 * Kernel pour transformer l'image RGB en niveaux de gris.
 */
__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }
}

/**
 * Kernel pour obtenir les contours à partir de l'image en niveaux de gris.
 */
__global__ void laplacian_gaussian( unsigned char * g, unsigned char * s, std::size_t cols, std::size_t rows )
{
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;

  if( i > 1 && i < cols && j > 1 && j < rows )
  {
      /*
    auto h =     g[ (j-1)*cols + i - 1 ] -     g[ (j-1)*cols + i + 1 ]
           + 2 * g[ (j  )*cols + i - 1 ] - 2 * g[ (j  )*cols + i + 1 ]
           +     g[ (j+1)*cols + i - 1 ] -     g[ (j+1)*cols + i + 1 ];

    auto v =     g[ (j-1)*cols + i - 1 ] -     g[ (j+1)*cols + i - 1 ]
           + 2 * g[ (j-1)*cols + i     ] - 2 * g[ (j+1)*cols + i     ]
           +     g[ (j-1)*cols + i + 1 ] -     g[ (j+1)*cols + i + 1 ];

    auto res = h*h + v*v;
    res = res > 65535 ? res = 65535 : res;

    s[ j * cols + i ] = sqrtf( res );
    */


      auto res =       g[((j - 2) * cols + i - 2) ] * 0 + g[((j - 2) * cols + i -1) ] * 0 +  g[((j - 2) * cols + i) ]* -1 + g[((j - 2) * cols + i +1 ) ] * 0 + g[((j - 2) * cols + i + 2) ] *0
                  +  g[((j - 1) * cols + i - 2) ] * 0 + g[((j - 1) * cols + i -1) ] * -1 +  g[((j - 1) * cols + i) ]* -2 + g[((j - 1) * cols + i +1 ) ] * -1 + g[((j - 1) * cols + i + 2) ] *0
                  +     g[((j) * cols + i - 2) ] * -1 + g[((j) * cols + i -1) ] * -2 +  g[((j) * cols + i) ]* 16 + g[((j) * cols + i +1 ) ] * -2 + g[((j) * cols + i + 2) ] * -1
                  +    g[((j + 1) * cols + i - 2) ] * 0 + g[((j +1) * cols + i -1) ] * -1 +  g[((j + 1) * cols + i) ]* -2 + g[((j + 1) * cols + i +1 ) ] * -1 + g[((j + 1) * cols + i + 2) ] *0
                  +    g[((j + 2) * cols + i - 2) ] * 0 + g[((j + 2) * cols + i -1) ] * 0 +  g[((j + 2) * cols + i) ]* -1 + g[((j + 2) * cols + i +1 ) ] * 0 + g[((j + 2) * cols + i + 2) ] *0;

      res = res > 255 ? 255 : res;
      res = res < 0 ? 0 : res;

      s[j * cols + i] = res;
  }
}


/**
 * Kernel pour obtenir les contours à partir de l'image en niveaux de gris, en utilisant la mémoire shared
 * pour limiter les accès à la mémoire globale.
 */
__global__ void laplacian_gaussian_shared( unsigned char * g, unsigned char * s, std::size_t cols, std::size_t rows )
{
  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  auto w = blockDim.x;
  auto h = blockDim.y;

  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  extern __shared__ unsigned char sh[];

  if( i < cols && j < rows )
  {
    sh[ lj * w + li ] = g[ j * cols + i ];
  }

  __syncthreads();

  if( i < cols -2 && j < rows-2 && li > 0 && li < (w-2) && lj > 0 && lj < (h-2) )
  {
      /*
    auto h =     sh[ (lj-1)*w + li - 1 ] -     sh[ (lj-1)*w + li + 1 ]
           + 2 * sh[ (lj  )*w + li - 1 ] - 2 * sh[ (lj  )*w + li + 1 ]
           +     sh[ (lj+1)*w + li - 1 ] -     sh[ (lj+1)*w + li + 1 ];

    auto v =     sh[ (lj-1)*w + li - 1 ] -     sh[ (lj+1)*w + li - 1 ]
           + 2 * sh[ (lj-1)*w + li     ] - 2 * sh[ (lj+1)*w + li     ]
           +     sh[ (lj-1)*w + li + 1 ] -     sh[ (lj+1)*w + li + 1 ];

    auto res = h*h + v*v;
    res = res > 65535 ? res = 65535 : res;

    s[ j * cols + i ] = sqrtf( res );
       */

      auto res =       sh[((lj - 2) * w + li - 2) ] * 0 + sh[((lj - 2) * w + li -1) ] * 0 +  sh[((lj - 2) * w + li) ]* -1 + sh[((lj - 2) * w + li +1 ) ] * 0 + sh[((lj - 2) * w + li + 2) ] *0
                       +  sh[((lj - 1) * w + li - 2) ] * 0 + sh[((lj - 1) * w + li -1) ] * -1 +  sh[((lj - 1) * w + li) ]* -2 + sh[((lj - 1) * w + li +1 ) ] * -1 + sh[((lj - 1) * w + li + 2) ] *0
                       +     sh[((lj) * w + li - 2) ] * -1 + sh[((lj) * w + li -1) ] * -2 +  sh[((lj) * w + li) ]* 16 + sh[((lj) * w + li +1 ) ] * -2 + sh[((lj) * w + li + 2) ] * -1
                       +    sh[((lj + 1) * w + li - 2) ] * 0 + sh[((lj +1) * w + li -1) ] * -1 +  sh[((lj + 1) * w + li) ]* -2 + sh[((lj + 1) * w + li +1 ) ] * -1 + sh[((lj + 1) * w + li + 2) ] *0
                       +    sh[((lj + 2) * w + li - 2) ] * 0 + sh[((lj + 2) * w + li -1) ] * 0 +  sh[((lj + 2) * w + li) ]* -1 + sh[((lj + 2) * w + li +1 ) ] * 0 + sh[((lj + 2) * w + li + 2) ] *0;

      res = res > 255 ? 255 : res;
      res = res < 0 ? 0 : res;

      s[j * cols + i] = res;


  }
}


/**
 * Kernel fusionnant le passage en niveaux de gris et la détection de contours.
 */
__global__ void grayscale_laplacian_gaussian_shared( unsigned char * rgb, unsigned char * s, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  auto w = blockDim.x;
  auto h = blockDim.y;

  extern __shared__ unsigned char sh[];

  if( i < cols && j < rows ) {
    sh[ lj * w + li ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }

  /**
   * Il faut synchroniser tous les warps (threads) du bloc pour être certain que le niveau de gris est calculé
   * par tous les threads du bloc avant de pouvoir accéder aux données des pixels voisins.
   */
  __syncthreads();
 
  if( i < cols -1 && j < rows-1 && li > 0 && li < (w-1) && lj > 0 && lj < (h-1) )
  {
      auto res =       sh[((lj - 2) * w + li - 2) ] * 0 + sh[((lj - 2) * w + li -1) ] * 0 +  sh[((lj - 2) * w + li) ]* -1 + sh[((lj - 2) * w + li +1 ) ] * 0 + sh[((lj - 2) * w + li + 2) ] *0
                       +  sh[((lj - 1) * w + li - 2) ] * 0 + sh[((lj - 1) * w + li -1) ] * -1 +  sh[((lj - 1) * w + li) ]* -2 + sh[((lj - 1) * w + li +1 ) ] * -1 + sh[((lj - 1) * w + li + 2) ] *0
                       +     sh[((lj) * w + li - 2) ] * -1 + sh[((lj) * w + li -1) ] * -2 +  sh[((lj) * w + li) ]* 16 + sh[((lj) * w + li +1 ) ] * -2 + sh[((lj) * w + li + 2) ] * -1
                       +    sh[((lj + 1) * w + li - 2) ] * 0 + sh[((lj +1) * w + li -1) ] * -1 +  sh[((lj + 1) * w + li) ]* -2 + sh[((lj + 1) * w + li +1 ) ] * -1 + sh[((lj + 1) * w + li + 2) ] *0
                       +    sh[((lj + 2) * w + li - 2) ] * 0 + sh[((lj + 2) * w + li -1) ] * 0 +  sh[((lj + 2) * w + li) ]* -1 + sh[((lj + 2) * w + li +1 ) ] * 0 + sh[((lj + 2) * w + li + 2) ] *0;

      res = res > 255 ? 255 : res;
      res = res < 0 ? 0 : res;

      s[j * cols + i] = res;
  }
}


int main()
{
  cv::Mat m_in = cv::imread("../images/in.jpg", cv::IMREAD_UNCHANGED );

  //auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  //std::vector< unsigned char > g( rows * cols );
  // Allocation de l'image de sortie en RAM côté CPU.
  unsigned char * g = nullptr;
  hipHostMalloc( &g, rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g );

  // Copie de l'image en entrée dans une mémoire dite "pinned" de manière à accélérer les transferts.
  // OpenCV alloue la mémoire en interne lors de la décompression de l'image donc soit sans doute avec
  // un malloc standard.
  unsigned char * rgb = nullptr;
  hipHostMalloc( &rgb, 3 * rows * cols );
  
  std::memcpy( rgb, m_in.data, 3 * rows * cols );

  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * s_d;

  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &g_d, rows * cols );
  hipMalloc( &s_d, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );

  dim3 block( 32, 4 );
  dim3 grid0( ( cols - 1) / block.x + 1 , ( rows - 1 ) / block.y + 1 );
  /**
   * Pour la version shared il faut faire superposer les blocs de 2 pixels
   * pour ne pas avoir de bandes non calculées autour des blocs
   * on crée donc plus de blocs.
   */
  dim3 grid1( ( cols - 1) / (block.x-2) + 1 , ( rows - 1 ) / (block.y-2) + 1 );
    
  hipEvent_t start, stop;

  hipEventCreate( &start );
  hipEventCreate( &stop );

  // Mesure du temps de calcul du kernel uniquement.
  hipEventRecord( start );

    /*
  // Version en 2 étapes.
  grayscale<<< grid0, block >>>( rgb_d, g_d, cols, rows );
  laplacian_gaussian<<< grid0, block >>>( g_d, s_d, cols, rows );
    */


  // Version en 2 étapes, Sobel avec mémoire shared.
  grayscale<<< grid0, block >>>( rgb_d, g_d, cols, rows );
  laplacian_gaussian_shared<<< grid1, block, block.x * block.y >>>( g_d, s_d, cols, rows );


  // Version fusionnée.
  //grayscale_laplacian_gaussian_shared<<< grid1, block, block.x * block.y >>>( rgb_d, s_d, cols, rows );

  hipEventRecord( stop );
  
  hipMemcpy( g, s_d, rows * cols, hipMemcpyDeviceToHost );

  hipEventSynchronize( stop );
  float duration;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "time=" << duration << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  cv::imwrite( "out.jpg", m_out );

  hipFree( rgb_d);
  hipFree( g_d);
  hipFree( s_d);

  hipHostFree( g );
  hipHostFree( rgb );
  
  return 0;
}
