/**
 * Fill a vector of 100 ints on the GPU with consecutive values.
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


__global__ void fill( int * v, std::size_t size )
{
  // Get the id of the thread ( 0 -> 99 ).
  auto tid = threadIdx.x;
  auto blockid = blockIdx.x;
  auto blockdim = blockDim.x;
  auto value = blockid*blockdim+tid;
  // Each thread fills a single element of the array. 
  v[ value ] = value;
}


int main()
{
  std::vector< int > v( 64 );

  int * v_d = nullptr;

  int nbDevices;
  hipGetDeviceCount(&nbDevices);

  printf("Number of devices : %d\n", nbDevices);


  // Allocate an array an the device.
  hipMalloc( &v_d, v.size() * sizeof( int ) );

  // Launch one block of 100 threads on the device.
  // In this block, threads are numbered from 0 to 99.
  fill<<< 8, 8 >>>( v_d, v.size() );

  // Copy data from the device memory to the host memory.
  hipMemcpy( v.data(), v_d, v.size() * sizeof( int ), hipMemcpyDeviceToHost );

  for( auto x: v )
  {
    std::cout << x << std::endl;
  }

  hipFree( v_d );

  return 0;
}
